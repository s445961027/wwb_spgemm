#include<hip/hip_runtime.h>
#include<stdio.h>

//不分块naive
__global__ void MatrixMul_naive(const float *A,const float *B,float *C,const int M,const int K,const int N)
{
    int y = blockDim.y * blockIdx.y + threadIdx.y;
    int x = blockDim.x * blockIdx.x + threadIdx.x;
    float sum = 0.0;
#pragma unroll
    for( int k = 0; k < K; k++){
        sum += A[y*K+k] * B[k*N+x]; 
    }
    C[y*N+x] = sum;

}


//分块，利用sharememory，每个thread计算一个element of C,每个块用内积的方式计算,非正方阵
template<int TILE_SIZE_M,int TILE_SIZE_K,int TILE_SIZE_N>
__global__ void MatrixMul1(const float *A,const float *B,float *C,const int M,const int K,const int N)
{

    __shared__ float local_A[TILE_SIZE_M][TILE_SIZE_K];
    __shared__ float local_B[TILE_SIZE_K][TILE_SIZE_N];

    // int baseY = TILE_SIZE_M * blockIdx.y;
    // int baseX = TILE_SIZE_N * blockIdx.x;
    float sum = 0.0;
    for(int tileid = 0; tileid < K/TILE_SIZE_K; tileid++)
    {
        //sharedmemory存在重复读取，太笨了，且存在bank冲突
        
        
        if(threadIdx.x < TILE_SIZE_K ){
            local_A[threadIdx.y][threadIdx.x] = A[(blockDim.y * blockIdx.y + threadIdx.y) * K + tileid * TILE_SIZE_K + threadIdx.x];
        }
        if(threadIdx.y < TILE_SIZE_K){
            local_B[threadIdx.y][threadIdx.x] = B[(tileid * TILE_SIZE_K + threadIdx.y) * N + blockDim.x * blockIdx.x + threadIdx.x];
        }
        __syncthreads();
    #pragma unroll
        for(int k = 0; k < TILE_SIZE_K; k++){
            sum += local_A[threadIdx.y][k] * local_B[k][threadIdx.x];
        }
        __syncthreads();
    }

    C[N * (blockDim.y * blockIdx.y + threadIdx.y) + blockDim.x * blockIdx.x + threadIdx.x] = sum;
     



}

//分块，每个thread计算多个element of C,向量内积方式  每个线程负责4 * 4个element of C，A100 每个SM有128个FFMA，4 * 4跑不满算力
template<int TILE_SIZE_M,int TILE_SIZE_K,int TILE_SIZE_N,int THREAD_BLOCK_M, int THREAD_BLOCK_N>
__global__ void MatrixMul2(const float *A,const float *B,float *C,const int M,const int K,const int N)
{
    __shared__ float local_A[TILE_SIZE_M][TILE_SIZE_K];
    __shared__ float local_B[TILE_SIZE_K][TILE_SIZE_N];
    float local_C[THREAD_BLOCK_M][THREAD_BLOCK_N];
    //在每个tile内每个线程负责的小块的起始位置
    for(int i = 0; i < THREAD_BLOCK_M; i++){
        for(int j = 0; j < THREAD_BLOCK_N; j++){
            local_C[i][j] = 0.0;
        }
    }
    // int In_Tile_startX = threadIdx.x * 4;
    // int In_Tile_startY = threadIdx.y * 4;
    
    for(int tileid = 0; tileid < K/TILE_SIZE_K; tileid++)
    {   
        //读取数据到共享内存 
        int index_A_Y = blockDim.y * THREAD_BLOCK_M * blockIdx.y + threadIdx.y * THREAD_BLOCK_M;
        int index_A_X = tileid * TILE_SIZE_K + threadIdx.x * THREAD_BLOCK_N;
        if(threadIdx.x*4 < TILE_SIZE_K){
            
            for(int j = 0; j < 4; j++)
            {   
                //

                // int index_A = (blockDim.y * THREAD_BLOCK_M * blockIdx.y + threadIdx.y * THREAD_BLOCK_M + j) * K + tileid * TILE_SIZE_K + threadIdx.x * 4;
                  
                local_A[threadIdx.y * 4 + j][threadIdx.x * 4] = A[(index_A_Y + j) * K + index_A_X];
                local_A[threadIdx.y * 4 + j][threadIdx.x * 4 + 1] = A[(index_A_Y + j) * K + index_A_X + 1];
                local_A[threadIdx.y * 4 + j][threadIdx.x * 4 + 2] = A[(index_A_Y + j) * K + index_A_X + 2];
                local_A[threadIdx.y * 4 + j][threadIdx.x * 4 + 3] = A[(index_A_Y + j) * K + index_A_X + 3];
            }
        }
        if(threadIdx.y*4 < TILE_SIZE_K)
        {   
            int index_B_Y = (tileid * TILE_SIZE_K + threadIdx.y * THREAD_BLOCK_M);
            int index_B_X = (blockDim.x * THREAD_BLOCK_N * blockIdx.x + threadIdx.x * THREAD_BLOCK_N);

            for(int j = 0; j < 4; j++)
            {
                
                local_B[threadIdx.y * 4 + j][threadIdx.x * 4] = B[(index_B_Y + j) * N + index_B_X];
                local_B[threadIdx.y * 4 + j][threadIdx.x * 4 + 1] = B[(index_B_Y + j) * N + index_B_X + 1];
                local_B[threadIdx.y * 4 + j][threadIdx.x * 4 + 2] = B[(index_B_Y + j) * N + index_B_X + 2];
                local_B[threadIdx.y * 4 + j][threadIdx.x * 4 + 3] = B[(index_B_Y + j) * N + index_B_X + 3];
            }
        }   
        __syncthreads();
        
        for(int k = 0; k < TILE_SIZE_K; k++)
        {
            for(int m = 0; m < THREAD_BLOCK_M; m++)
            {
                for(int n = 0; n < THREAD_BLOCK_N; n++)
                {
                    //外积   
                    local_C[m][n] += local_A[threadIdx.y * THREAD_BLOCK_M + m][k]*local_B[k][threadIdx.x * THREAD_BLOCK_N + n];
                }
            }
        }
        __syncthreads();
    }
    //结果写回主存,可以做float4向量化写回
    int y = (blockDim.y * blockIdx.y + threadIdx.y) * THREAD_BLOCK_M;
    int x = (blockDim.x * blockIdx.x + threadIdx.x) * THREAD_BLOCK_N;
    for(int thread_y = 0; thread_y < THREAD_BLOCK_M; thread_y++)
    {
        for(int thread_x = 0; thread_x < THREAD_BLOCK_N; thread_x++)
        {
            C[(y+thread_y) * N + x + thread_x] = local_C[thread_y][thread_x];
        }
    }

    
}


//分块，每个thread计算多个element，向量外积方式, 添加sharedmem to register层次
template<int TILE_SIZE_M,int TILE_SIZE_K,int TILE_SIZE_N,int THREAD_BLOCK_M,int THREAD_BLOCK_N>
__global__ void MatrixMul3(const float *A,const float *B,float *C,const int M,const int K,const int N){
    
    __shared__ float local_A[TILE_SIZE_M][TILE_SIZE_K];
    __shared__ float local_B[TILE_SIZE_K][TILE_SIZE_N];
    float local_A_col[THREAD_BLOCK_M];
    float local_B_row[THREAD_BLOCK_N];
    float local_C[THREAD_BLOCK_M][THREAD_BLOCK_N];
    //在每个tile内每个线程负责的小块的起始位置
    for(int i = 0; i < THREAD_BLOCK_M; i++){
        for(int j = 0; j < THREAD_BLOCK_N; j++){
            local_C[i][j] = 0.0;
        }
    }
    // int In_Tile_startX = threadIdx.x * 4;
    // int In_Tile_startY = threadIdx.y * 4;
    
    for(int tileid = 0; tileid < K/TILE_SIZE_K; tileid++)
    {   
        //读取数据到共享内存 
        int index_A_Y = blockDim.y * THREAD_BLOCK_M * blockIdx.y + threadIdx.y * THREAD_BLOCK_M;
        int index_A_X = tileid * TILE_SIZE_K + threadIdx.x * THREAD_BLOCK_N;
        if(threadIdx.x*4 < TILE_SIZE_K){
            
            for(int j = 0; j < 4; j++)
            {   
                //

                // int index_A = (blockDim.y * THREAD_BLOCK_M * blockIdx.y + threadIdx.y * THREAD_BLOCK_M + j) * K + tileid * TILE_SIZE_K + threadIdx.x * 4;
                  
                local_A[threadIdx.y * 4 + j][threadIdx.x * 4] = A[(index_A_Y + j) * K + index_A_X];
                local_A[threadIdx.y * 4 + j][threadIdx.x * 4 + 1] = A[(index_A_Y + j) * K + index_A_X + 1];
                local_A[threadIdx.y * 4 + j][threadIdx.x * 4 + 2] = A[(index_A_Y + j) * K + index_A_X + 2];
                local_A[threadIdx.y * 4 + j][threadIdx.x * 4 + 3] = A[(index_A_Y + j) * K + index_A_X + 3];
            }
        }
        if(threadIdx.y*4 < TILE_SIZE_K)
        {   
            int index_B_Y = (tileid * TILE_SIZE_K + threadIdx.y * THREAD_BLOCK_M);
            int index_B_X = (blockDim.x * THREAD_BLOCK_N * blockIdx.x + threadIdx.x * THREAD_BLOCK_N);

            for(int j = 0; j < 4; j++)
            {
                
                local_B[threadIdx.y * 4 + j][threadIdx.x * 4] = B[(index_B_Y + j) * N + index_B_X];
                local_B[threadIdx.y * 4 + j][threadIdx.x * 4 + 1] = B[(index_B_Y + j) * N + index_B_X + 1];
                local_B[threadIdx.y * 4 + j][threadIdx.x * 4 + 2] = B[(index_B_Y + j) * N + index_B_X + 2];
                local_B[threadIdx.y * 4 + j][threadIdx.x * 4 + 3] = B[(index_B_Y + j) * N + index_B_X + 3];
            }
        }   
        __syncthreads();
        
        for(int k = 0; k < TILE_SIZE_K; k++)
        {   
            for(int mm = 0; mm < THREAD_BLOCK_M; mm++){
                //二路bank冲突
                local_A_col[mm] = local_A[threadIdx.y*4 + mm][k];
            }
            for(int nn = 0; nn < THREAD_BLOCK_N; nn++){
                //四路bank冲突
                local_B_row[nn] = local_B[k][threadIdx.x*4 + nn];
            }
            
            for(int m = 0; m < THREAD_BLOCK_M; m++)
            {
                for(int n = 0; n < THREAD_BLOCK_N; n++)
                {
                    //外积   
                    // local_C[m][n] += local_A[threadIdx.y * THREAD_BLOCK_M + m][k]*local_B[k][threadIdx.x * THREAD_BLOCK_N + n];
                    local_C[m][n] += local_A_col[m] *local_B_row[n];
                }
            }
        }
        __syncthreads();
    }
    //结果写回主存,可以做float4向量化写回
    int y = (blockDim.y * blockIdx.y + threadIdx.y) * THREAD_BLOCK_M;
    int x = (blockDim.x * blockIdx.x + threadIdx.x) * THREAD_BLOCK_N;
    for(int thread_y = 0; thread_y < THREAD_BLOCK_M; thread_y++)
    {
        for(int thread_x = 0; thread_x < THREAD_BLOCK_N; thread_x++)
        {
            C[(y+thread_y) * N + x + thread_x] = local_C[thread_y][thread_x];
        }
    }


}

//